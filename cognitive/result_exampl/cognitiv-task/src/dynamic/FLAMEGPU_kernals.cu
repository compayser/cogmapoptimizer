#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


#ifndef _FLAMEGPU_KERNELS_H_
#define _FLAMEGPU_KERNELS_H_

#include "header.h"


/* Agent count constants */

__constant__ int d_xmachine_memory_vertice_count;

/* Agent state count constants */

__constant__ int d_xmachine_memory_vertice_default_count;


/* Message constants */

/* send_local Message variables */
/* Non partitioned, spatial partitioned and on-graph partitioned message variables  */
__constant__ int d_message_send_local_count;         /**< message list counter*/
__constant__ int d_message_send_local_output_type;   /**< message output type (single or optional)*/

	

/* Graph Constants */


/* Graph device array pointer(s) */


/* Graph host array pointer(s) */

    
//include each function file

#include "functions.c"
    
/* Texture bindings */

    
#define WRAP(x,m) (((x)<m)?(x):(x%m)) /**< Simple wrap */
#define sWRAP(x,m) (((x)<m)?(((x)<0)?(m+(x)):(x)):(m-(x))) /**<signed integer wrap (no modulus) for negatives where 2m > |x| > m */

//PADDING WILL ONLY AVOID SM CONFLICTS FOR 32BIT
//SM_OFFSET REQUIRED AS FERMI STARTS INDEXING MEMORY FROM LOCATION 0 (i.e. NULL)??
__constant__ int d_SM_START;
__constant__ int d_PADDING;

//SM addressing macro to avoid conflicts (32 bit only)
#define SHARE_INDEX(i, s) ((((s) + d_PADDING)* (i))+d_SM_START) /**<offset struct size by padding to avoid bank conflicts */

//if doubel support is needed then define the following function which requires sm_13 or later
#ifdef _DOUBLE_SUPPORT_REQUIRED_
__inline__ __device__ double tex1DfetchDouble(texture<int2, 1, hipReadModeElementType> tex, int i)
{
	int2 v = tex1Dfetch(tex, i);
  //IF YOU HAVE AN ERROR HERE THEN YOU ARE USING DOUBLE VALUES IN AGENT MEMORY AND NOT COMPILING FOR DOUBLE SUPPORTED HARDWARE
  //To compile for double supported hardware change the CUDA Build rule property "Use sm_13 Architecture (double support)" on the CUDA-Specific Propert Page of the CUDA Build Rule for simulation.cu
	return __hiloint2double(v.y, v.x);
}
#endif

/* Helper functions */
/** next_cell
 * Function used for finding the next cell when using spatial partitioning
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1,1
 */
__device__ bool next_cell3D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	if (relative_cell->z < 1)
	{
		relative_cell->z++;
		return true;
	}
	relative_cell->z = -1;
	
	return false;
}

/** next_cell2D
 * Function used for finding the next cell when using spatial partitioning. Z component is ignored
 * Upddates the relative cell variable which can have value of -1, 0 or +1
 * @param relative_cell pointer to the relative cell position
 * @return boolean if there is a next cell. True unless relative_Cell value was 1,1
 */
__device__ bool next_cell2D(glm::ivec3* relative_cell)
{
	if (relative_cell->x < 1)
	{
		relative_cell->x++;
		return true;
	}
	relative_cell->x = -1;

	if (relative_cell->y < 1)
	{
		relative_cell->y++;
		return true;
	}
	relative_cell->y = -1;
	
	return false;
}


////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created vertice agent functions */

/** reset_vertice_scan_input
 * vertice agent reset scan input function
 * @param agents The xmachine_memory_vertice_list agent list
 */
__global__ void reset_vertice_scan_input(xmachine_memory_vertice_list* agents){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	agents->_position[index] = 0;
	agents->_scan_input[index] = 0;
}



/** scatter_vertice_Agents
 * vertice scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_vertice_list agent list destination
 * @param agents_src xmachine_memory_vertice_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void scatter_vertice_Agents(xmachine_memory_vertice_list* agents_dst, xmachine_memory_vertice_list* agents_src, int dst_agent_count, int number_to_scatter){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = agents_src->_scan_input[index];

	//if optional message is to be written. 
	//must check agent is within number to scatter as unused threads may have scan input = 1
	if ((_scan_input == 1)&&(index < number_to_scatter)){
		int output_index = agents_src->_position[index] + dst_agent_count;

		//AoS - xmachine_message_location Un-Coalesced scattered memory write     
        agents_dst->_position[output_index] = output_index;        
		agents_dst->id[output_index] = agents_src->id[index];        
		agents_dst->value[output_index] = agents_src->value[index];        
		agents_dst->add_value[output_index] = agents_src->add_value[index];        
		agents_dst->previous[output_index] = agents_src->previous[index];        
		agents_dst->max_lag[output_index] = agents_src->max_lag[index];        
		agents_dst->current_lag[output_index] = agents_src->current_lag[index];
	    for (int i=0; i<17; i++){
	      agents_dst->edges[(i*xmachine_memory_vertice_MAX)+output_index] = agents_src->edges[(i*xmachine_memory_vertice_MAX)+index];
	    }        
		agents_dst->min[output_index] = agents_src->min[index];        
		agents_dst->max[output_index] = agents_src->max[index];        
		agents_dst->need_test[output_index] = agents_src->need_test[index];        
		agents_dst->correct[output_index] = agents_src->correct[index];
	}
}

/** append_vertice_Agents
 * vertice scatter agents function (used after agent birth/death)
 * @param agents_dst xmachine_memory_vertice_list agent list destination
 * @param agents_src xmachine_memory_vertice_list agent list source
 * @param dst_agent_count index to start scattering agents from
 */
__global__ void append_vertice_Agents(xmachine_memory_vertice_list* agents_dst, xmachine_memory_vertice_list* agents_src, int dst_agent_count, int number_to_append){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//must check agent is within number to append as unused threads may have scan input = 1
    if (index < number_to_append){
	    int output_index = index + dst_agent_count;

	    //AoS - xmachine_message_location Un-Coalesced scattered memory write
	    agents_dst->_position[output_index] = output_index;
	    agents_dst->id[output_index] = agents_src->id[index];
	    agents_dst->value[output_index] = agents_src->value[index];
	    agents_dst->add_value[output_index] = agents_src->add_value[index];
	    agents_dst->previous[output_index] = agents_src->previous[index];
	    agents_dst->max_lag[output_index] = agents_src->max_lag[index];
	    agents_dst->current_lag[output_index] = agents_src->current_lag[index];
	    for (int i=0; i<17; i++){
	      agents_dst->edges[(i*xmachine_memory_vertice_MAX)+output_index] = agents_src->edges[(i*xmachine_memory_vertice_MAX)+index];
	    }
	    agents_dst->min[output_index] = agents_src->min[index];
	    agents_dst->max[output_index] = agents_src->max[index];
	    agents_dst->need_test[output_index] = agents_src->need_test[index];
	    agents_dst->correct[output_index] = agents_src->correct[index];
    }
}

/** add_vertice_agent
 * Continuous vertice agent add agent function writes agent data to agent swap
 * @param agents xmachine_memory_vertice_list to add agents to 
 * @param id agent variable of type int
 * @param value agent variable of type float
 * @param add_value agent variable of type float
 * @param previous agent variable of type float
 * @param max_lag agent variable of type int
 * @param current_lag agent variable of type int
 * @param edges agent variable of type float
 * @param min agent variable of type float
 * @param max agent variable of type float
 * @param need_test agent variable of type int
 * @param correct agent variable of type int
 */
template <int AGENT_TYPE>
__device__ void add_vertice_agent(xmachine_memory_vertice_list* agents, int id, float value, float add_value, float previous, int max_lag, int current_lag, float min, float max, int need_test, int correct){
	
	int index;
    
    //calculate the agents index in global agent list (depends on agent type)
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x* gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x*blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y*blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y* width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	//for prefix sum
	agents->_position[index] = 0;
	agents->_scan_input[index] = 1;

	//write data to new buffer
	agents->id[index] = id;
	agents->value[index] = value;
	agents->add_value[index] = add_value;
	agents->previous[index] = previous;
	agents->max_lag[index] = max_lag;
	agents->current_lag[index] = current_lag;
	agents->min[index] = min;
	agents->max[index] = max;
	agents->need_test[index] = need_test;
	agents->correct[index] = correct;

}

//non templated version assumes DISCRETE_2D but works also for CONTINUOUS
__device__ void add_vertice_agent(xmachine_memory_vertice_list* agents, int id, float value, float add_value, float previous, int max_lag, int current_lag, float min, float max, int need_test, int correct){
    add_vertice_agent<DISCRETE_2D>(agents, id, value, add_value, previous, max_lag, current_lag, min, max, need_test, correct);
}

/** reorder_vertice_agents
 * Continuous vertice agent areorder function used after key value pairs have been sorted
 * @param values sorted index values
 * @param unordered_agents list of unordered agents
 * @ param ordered_agents list used to output ordered agents
 */
__global__ void reorder_vertice_agents(unsigned int* values, xmachine_memory_vertice_list* unordered_agents, xmachine_memory_vertice_list* ordered_agents)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	uint old_pos = values[index];

	//reorder agent data
	ordered_agents->id[index] = unordered_agents->id[old_pos];
	ordered_agents->value[index] = unordered_agents->value[old_pos];
	ordered_agents->add_value[index] = unordered_agents->add_value[old_pos];
	ordered_agents->previous[index] = unordered_agents->previous[old_pos];
	ordered_agents->max_lag[index] = unordered_agents->max_lag[old_pos];
	ordered_agents->current_lag[index] = unordered_agents->current_lag[old_pos];
	for (int i=0; i<17; i++){
	  ordered_agents->edges[(i*xmachine_memory_vertice_MAX)+index] = unordered_agents->edges[(i*xmachine_memory_vertice_MAX)+old_pos];
	}
	ordered_agents->min[index] = unordered_agents->min[old_pos];
	ordered_agents->max[index] = unordered_agents->max[old_pos];
	ordered_agents->need_test[index] = unordered_agents->need_test[old_pos];
	ordered_agents->correct[index] = unordered_agents->correct[old_pos];
}

/** get_vertice_agent_array_value
 *  Template function for accessing vertice agent array memory variables. Assumes array points to the first element of the agents array values (offset by agent index)
 *  @param array Agent memory array
 *  @param index to lookup
 *  @return return value
 */
template<typename T>
__FLAME_GPU_FUNC__ T get_vertice_agent_array_value(T *array, uint index){
	// Null check for out of bounds agents (brute force communication. )
	if(array != nullptr){
	    return array[index*xmachine_memory_vertice_MAX];
    } else {
    	// Return the default value for this data type 
	    return 0;
    }
}

/** set_vertice_agent_array_value
 *  Template function for setting vertice agent array memory variables. Assumes array points to the first element of the agents array values (offset by agent index)
 *  @param array Agent memory array
 *  @param index to lookup
 *  @param return value
 */
template<typename T>
__FLAME_GPU_FUNC__ void set_vertice_agent_array_value(T *array, uint index, T value){
	// Null check for out of bounds agents (brute force communication. )
	if(array != nullptr){
	    array[index*xmachine_memory_vertice_MAX] = value;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created send_local message functions */


/** add_send_local_message
 * Add non partitioned or spatially partitioned send_local message
 * @param messages xmachine_message_send_local_list message list to add too
 * @param from_id agent variable of type int
 * @param value agent variable of type float
 */
__device__ void add_send_local_message(xmachine_message_send_local_list* messages, int from_id, float value){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x + d_message_send_local_count;

	int _position;
	int _scan_input;

	//decide output position
	if(d_message_send_local_output_type == single_message){
		_position = index; //same as agent position
		_scan_input = 0;
	}else if (d_message_send_local_output_type == optional_message){
		_position = 0;	   //to be calculated using Prefix sum
		_scan_input = 1;
	}

	//AoS - xmachine_message_send_local Coalesced memory write
	messages->_scan_input[index] = _scan_input;	
	messages->_position[index] = _position;
	messages->from_id[index] = from_id;
	messages->value[index] = value;

}

/**
 * Scatter non partitioned or spatially partitioned send_local message (for optional messages)
 * @param messages scatter_optional_send_local_messages Sparse xmachine_message_send_local_list message list
 * @param message_swap temp xmachine_message_send_local_list message list to scatter sparse messages to
 */
__global__ void scatter_optional_send_local_messages(xmachine_message_send_local_list* messages, xmachine_message_send_local_list* messages_swap){
	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	int _scan_input = messages_swap->_scan_input[index];

	//if optional message is to be written
	if (_scan_input == 1){
		int output_index = messages_swap->_position[index] + d_message_send_local_count;

		//AoS - xmachine_message_send_local Un-Coalesced scattered memory write
		messages->_position[output_index] = output_index;
		messages->from_id[output_index] = messages_swap->from_id[index];
		messages->value[output_index] = messages_swap->value[index];				
	}
}

/** reset_send_local_swaps
 * Reset non partitioned or spatially partitioned send_local message swaps (for scattering optional messages)
 * @param message_swap message list to reset _position and _scan_input values back to 0
 */
__global__ void reset_send_local_swaps(xmachine_message_send_local_list* messages_swap){

	//global thread index
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;

	messages_swap->_position[index] = 0;
	messages_swap->_scan_input[index] = 0;
}

/* Message functions */

__device__ xmachine_message_send_local* get_first_send_local_message(xmachine_message_send_local_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = (ceil((float)d_message_send_local_count/ blockDim.x)* blockDim.x);

	//if no messages then return a null pointer (false)
	if (wrap_size == 0)
		return nullptr;

	//global thread index
	int global_index = (blockIdx.x*blockDim.x) + threadIdx.x;

	//global thread index
	int index = WRAP(global_index, wrap_size);

	//SoA to AoS - xmachine_message_send_local Coalesced memory read
	xmachine_message_send_local temp_message;
	temp_message._position = messages->_position[index];
	temp_message.from_id = messages->from_id[index];
	temp_message.value = messages->value[index];

	//AoS to shared memory
	int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_send_local));
	xmachine_message_send_local* sm_message = ((xmachine_message_send_local*)&message_share[message_index]);
	sm_message[0] = temp_message;

	__syncthreads();

  //HACK FOR 64 bit addressing issue in sm
	return ((xmachine_message_send_local*)&message_share[d_SM_START]);
}

__device__ xmachine_message_send_local* get_next_send_local_message(xmachine_message_send_local* message, xmachine_message_send_local_list* messages){

	extern __shared__ int sm_data [];
	char* message_share = (char*)&sm_data[0];
	
	//wrap size is the number of tiles required to load all messages
	int wrap_size = ceil((float)d_message_send_local_count/ blockDim.x)*blockDim.x;

	int i = WRAP((message->_position + 1),wrap_size);

	//If end of messages (last message not multiple of gridsize) go to 0 index
	if (i >= d_message_send_local_count)
		i = 0;

	//Check if back to start position of first message
	if (i == WRAP((blockDim.x* blockIdx.x), wrap_size))
		return nullptr;

	int tile = floor((float)i/(blockDim.x)); //tile is round down position over blockDim
	i = i % blockDim.x;						 //mod i for shared memory index

	//if count == Block Size load next tile int shared memory values
	if (i == 0){
		__syncthreads();					//make sure we don't change shared memory until all threads are here (important for emu-debug mode)
		
		//SoA to AoS - xmachine_message_send_local Coalesced memory read
		int index = (tile* blockDim.x) + threadIdx.x;
		xmachine_message_send_local temp_message;
		temp_message._position = messages->_position[index];
		temp_message.from_id = messages->from_id[index];
		temp_message.value = messages->value[index];

		//AoS to shared memory
		int message_index = SHARE_INDEX(threadIdx.y*blockDim.x+threadIdx.x, sizeof(xmachine_message_send_local));
		xmachine_message_send_local* sm_message = ((xmachine_message_send_local*)&message_share[message_index]);
		sm_message[0] = temp_message;

		__syncthreads();					//make sure we don't start returning messages until all threads have updated shared memory
	}

	int message_index = SHARE_INDEX(i, sizeof(xmachine_message_send_local));
	return ((xmachine_message_send_local*)&message_share[message_index]);
}

	
/////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Dynamically created GPU kernels  */



/**
 *
 */
__global__ void GPUFLAME_send_message(xmachine_memory_vertice_list* agents, xmachine_message_send_local_list* send_local_messages, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    //For agents not using non partitioned message input check the agent bounds
    if (index >= d_xmachine_memory_vertice_count)
        return;
    

	//SoA to AoS - xmachine_memory_send_message Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_vertice agent;
    
    // Thread bounds already checked, but the agent function will still execute. load default values?
	
	agent.id = agents->id[index];
	agent.value = agents->value[index];
	agent.add_value = agents->add_value[index];
	agent.previous = agents->previous[index];
	agent.max_lag = agents->max_lag[index];
	agent.current_lag = agents->current_lag[index];
    agent.edges = &(agents->edges[index]);
	agent.min = agents->min[index];
	agent.max = agents->max[index];
	agent.need_test = agents->need_test[index];
	agent.correct = agents->correct[index];

	//FLAME function call
	int dead = !send_message(&agent, send_local_messages	, rand48);
	

	//continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_send_message Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->value[index] = agent.value;
	agents->add_value[index] = agent.add_value;
	agents->previous[index] = agent.previous;
	agents->max_lag[index] = agent.max_lag;
	agents->current_lag[index] = agent.current_lag;
	agents->min[index] = agent.min;
	agents->max[index] = agent.max;
	agents->need_test[index] = agent.need_test;
	agents->correct[index] = agent.correct;
}

/**
 *
 */
__global__ void GPUFLAME_read_message(xmachine_memory_vertice_list* agents, xmachine_message_send_local_list* send_local_messages, RNG_rand48* rand48){
	
	//continuous agent: index is agent position in 1D agent list
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  
    
    //No partitioned input requires threads to be launched beyond the agent count to ensure full block sizes
    

	//SoA to AoS - xmachine_memory_read_message Coalesced memory read (arrays point to first item for agent index)
	xmachine_memory_vertice agent;
    //No partitioned input may launch more threads than required - only load agent data within bounds. 
    if (index < d_xmachine_memory_vertice_count){
    
	agent.id = agents->id[index];
	agent.value = agents->value[index];
	agent.add_value = agents->add_value[index];
	agent.previous = agents->previous[index];
	agent.max_lag = agents->max_lag[index];
	agent.current_lag = agents->current_lag[index];
    agent.edges = &(agents->edges[index]);
	agent.min = agents->min[index];
	agent.max = agents->max[index];
	agent.need_test = agents->need_test[index];
	agent.correct = agents->correct[index];
	} else {
	
	agent.id = 0;
	agent.value = 0;
	agent.add_value = 0;
	agent.previous = 0;
	agent.max_lag = 0;
	agent.current_lag = 0;
    agent.edges = nullptr;
	agent.min = 0;
	agent.max = 0;
	agent.need_test = 0;
	agent.correct = 0;
	}

	//FLAME function call
	int dead = !read_message(&agent, send_local_messages, rand48);
	

	
    //No partitioned input may launch more threads than required - only write agent data within bounds. 
    if (index < d_xmachine_memory_vertice_count){
    //continuous agent: set reallocation flag
	agents->_scan_input[index]  = dead; 

	//AoS to SoA - xmachine_memory_read_message Coalesced memory write (ignore arrays)
	agents->id[index] = agent.id;
	agents->value[index] = agent.value;
	agents->add_value[index] = agent.add_value;
	agents->previous[index] = agent.previous;
	agents->max_lag[index] = agent.max_lag;
	agents->current_lag[index] = agent.current_lag;
	agents->min[index] = agent.min;
	agents->max[index] = agent.max;
	agents->need_test[index] = agent.need_test;
	agents->correct[index] = agent.correct;
	}
}

	

/* Agent ID Generation functions implemented in simulation.cu and FLAMEGPU_kernals.cu*/

__FLAME_GPU_HOST_FUNC__ __FLAME_GPU_FUNC__ int generate_vertice_id(){
#if defined(__CUDA_ARCH__)
	// On the device, use atomicAdd to increment the ID, wrapping at overflow. Does not use atomicInc which only supports unsigned
	int new_id = atomicAdd(&d_current_value_generate_vertice_id, 1);
	return new_id;	
#else
	// On the host, get the current value to be returned and increment the host value.
	int new_id = h_current_value_generate_vertice_id;
	h_current_value_generate_vertice_id++; 
	return new_id;
#endif
}

	
/* Graph utility functions */



/////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/* Rand48 functions */

__device__ static glm::uvec2 RNG_rand48_iterate_single(glm::uvec2 Xn, glm::uvec2 A, glm::uvec2 C)
{
	unsigned int R0, R1;

	// low 24-bit multiplication
	const unsigned int lo00 = __umul24(Xn.x, A.x);
	const unsigned int hi00 = __umulhi(Xn.x, A.x);

	// 24bit distribution of 32bit multiplication results
	R0 = (lo00 & 0xFFFFFF);
	R1 = (lo00 >> 24) | (hi00 << 8);

	R0 += C.x; R1 += C.y;

	// transfer overflows
	R1 += (R0 >> 24);
	R0 &= 0xFFFFFF;

	// cross-terms, low/hi 24-bit multiplication
	R1 += __umul24(Xn.y, A.x);
	R1 += __umul24(Xn.x, A.y);

	R1 &= 0xFFFFFF;

	return glm::uvec2(R0, R1);
}

//Templated function
template <int AGENT_TYPE>
__device__ float rnd(RNG_rand48* rand48){

	int index;
	
	//calculate the agents index in global agent list
	if (AGENT_TYPE == DISCRETE_2D){
		int width = (blockDim.x * gridDim.x);
		glm::ivec2 global_position;
		global_position.x = (blockIdx.x * blockDim.x) + threadIdx.x;
		global_position.y = (blockIdx.y * blockDim.y) + threadIdx.y;
		index = global_position.x + (global_position.y * width);
	}else//AGENT_TYPE == CONTINOUS
		index = threadIdx.x + blockIdx.x*blockDim.x;

	glm::uvec2 state = rand48->seeds[index];
	glm::uvec2 A = rand48->A;
	glm::uvec2 C = rand48->C;

	int rand = ( state.x >> 17 ) | ( state.y << 7);

	// this actually iterates the RNG
	state = RNG_rand48_iterate_single(state, A, C);

	rand48->seeds[index] = state;

	return (float)rand/2147483647;
}

__device__ float rnd(RNG_rand48* rand48){
	return rnd<DISCRETE_2D>(rand48);
}

#endif //_FLAMEGPU_KERNELS_H_
