#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <limits.h>
#include <algorithm>
#include <string>
#include <vector>



#ifdef _WIN32
#define strtok_r strtok_s
#endif

// include header
#include "header.h"

glm::vec3 agent_maximum;
glm::vec3 agent_minimum;

int fpgu_strtol(const char* str){
    return (int)strtol(str, NULL, 0);
}

unsigned int fpgu_strtoul(const char* str){
    return (unsigned int)strtoul(str, NULL, 0);
}

long long int fpgu_strtoll(const char* str){
    return strtoll(str, NULL, 0);
}

unsigned long long int fpgu_strtoull(const char* str){
    return strtoull(str, NULL, 0);
}

double fpgu_strtod(const char* str){
    return strtod(str, NULL);
}

float fgpu_atof(const char* str){
    return (float)atof(str);
}


//templated class function to read array inputs from supported types
template <class T>
void readArrayInput( T (*parseFunc)(const char*), char* buffer, T *array, unsigned int expected_items, const char * agent_name, const char * variable_name){
    unsigned int i = 0;
    const char s[2] = ",";
    char * token;
    char * end_str;

    token = strtok_r(buffer, s, &end_str);
    while (token != NULL){
        if (i>=expected_items){
            fprintf(stderr, "Error: variable array %s->%s has too many items (%d), expected %d!\n", agent_name, variable_name, i, expected_items);
            exit(EXIT_FAILURE);
        }
        
        array[i++] = (T)parseFunc(token);
        
        token = strtok_r(NULL, s, &end_str);
    }
    #if ! defined(SUPPRESS_VARIABLE_ARRAY_ELEMENT_WARNING)
    if (i != expected_items){
        fprintf(stderr, "Warning: variable array %s->%s has %d items, expected %d!\n", agent_name, variable_name, i, expected_items);
        
    }
    #endif
}

//templated class function to read array inputs from supported types
template <class T, class BASE_T, unsigned int D>
void readArrayInputVectorType( BASE_T (*parseFunc)(const char*), char* buffer, T *array, unsigned int expected_items, const char * agent_name, const char * variable_name){
    unsigned int i = 0;
    const char s[2] = "|";
    char * token;
    char * end_str;

    token = strtok_r(buffer, s, &end_str);
    while (token != NULL){
        if (i>=expected_items){
            fprintf(stderr, "Error: variable array of vectors %s->%s has too many items (%d), expected %d!\n", agent_name, variable_name, i, expected_items);
        }
        
        //read vector type as an array
        T vec;
        readArrayInput<BASE_T>(parseFunc, token, (BASE_T*) &vec, D);
        array[i++] = vec;
        
        token = strtok_r(NULL, s, &end_str);
    }
    #if ! defined(SUPPRESS_VARIABLE_ARRAY_ELEMENT_WARNING)
    if (i != expected_items){
        fprintf(stderr, "Warning: variable array of vectors %s->%s has %d items, expected %d!\n", agent_name, variable_name, i, expected_items);
        
    }
    #endif
}

void saveIterationData(char* outputpath, int iteration_number, xmachine_memory_vertice_list* h_vertices_default, xmachine_memory_vertice_list* d_vertices_default, int h_xmachine_memory_vertice_default_count)
{
    PROFILE_SCOPED_RANGE("saveIterationData");
	hipError_t cudaStatus;
	
	//Device to host memory transfer
	
	cudaStatus = hipMemcpy( h_vertices_default, d_vertices_default, sizeof(xmachine_memory_vertice_list), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr,"Error Copying vertice Agent default State Memory from GPU: %s\n", hipGetErrorString(cudaStatus));
		exit(cudaStatus);
	}
	
	/* Pointer to file */
	FILE *file;
	char data[100];

	sprintf(data, "%s%i.xml", outputpath, iteration_number);
	//printf("Writing iteration %i data to %s\n", iteration_number, data);
	file = fopen(data, "w");
    if(file == nullptr){
        printf("Error: Could not open file `%s` for output. Aborting.\n", data);
        exit(EXIT_FAILURE);
    }
    fputs("<states>\n<itno>", file);
    sprintf(data, "%i", iteration_number);
    fputs(data, file);
    fputs("</itno>\n", file);
    fputs("<environment>\n" , file);
    
    fputs("\t<VERTICES_COUNT>", file);
    sprintf(data, "%d", (*get_VERTICES_COUNT()));
    fputs(data, file);
    fputs("</VERTICES_COUNT>\n", file);
	fputs("</environment>\n" , file);

	//Write each vertice agent to xml
	for (int i=0; i<h_xmachine_memory_vertice_default_count; i++){
		fputs("<xagent>\n" , file);
		fputs("<name>vertice</name>\n", file);
        
		fputs("<id>", file);
        sprintf(data, "%d", h_vertices_default->id[i]);
		fputs(data, file);
		fputs("</id>\n", file);
        
		fputs("<value>", file);
        sprintf(data, "%f", h_vertices_default->value[i]);
		fputs(data, file);
		fputs("</value>\n", file);
        
		fputs("<add_value>", file);
        sprintf(data, "%f", h_vertices_default->add_value[i]);
		fputs(data, file);
		fputs("</add_value>\n", file);
        
		fputs("<previous>", file);
        sprintf(data, "%f", h_vertices_default->previous[i]);
		fputs(data, file);
		fputs("</previous>\n", file);
        
		fputs("<max_lag>", file);
        sprintf(data, "%d", h_vertices_default->max_lag[i]);
		fputs(data, file);
		fputs("</max_lag>\n", file);
        
		fputs("<current_lag>", file);
        sprintf(data, "%d", h_vertices_default->current_lag[i]);
		fputs(data, file);
		fputs("</current_lag>\n", file);
        
		fputs("<edges>", file);
        for (int j=0;j<17;j++){
            fprintf(file, "%f", h_vertices_default->edges[(j*xmachine_memory_vertice_MAX)+i]);
            if(j!=(17-1))
                fprintf(file, ",");
        }
		fputs("</edges>\n", file);
        
		fputs("<min>", file);
        sprintf(data, "%f", h_vertices_default->min[i]);
		fputs(data, file);
		fputs("</min>\n", file);
        
		fputs("<max>", file);
        sprintf(data, "%f", h_vertices_default->max[i]);
		fputs(data, file);
		fputs("</max>\n", file);
        
		fputs("<need_test>", file);
        sprintf(data, "%d", h_vertices_default->need_test[i]);
		fputs(data, file);
		fputs("</need_test>\n", file);
        
		fputs("<correct>", file);
        sprintf(data, "%d", h_vertices_default->correct[i]);
		fputs(data, file);
		fputs("</correct>\n", file);
        
		fputs("</xagent>\n", file);
	}
	
	

	fputs("</states>\n" , file);
	
	/* Close the file */
	fclose(file);

}

void initEnvVars()
{
PROFILE_SCOPED_RANGE("initEnvVars");

    int t_VERTICES_COUNT = (int)17;
    set_VERTICES_COUNT(&t_VERTICES_COUNT);
}

void readInitialStates(char* inputpath, xmachine_memory_vertice_list* h_vertices, int* h_xmachine_memory_vertice_count)
{
    PROFILE_SCOPED_RANGE("readInitialStates");

	int temp = 0;
	int* itno = &temp;

	/* Pointer to file */
	FILE *file;
	/* Char and char buffer for reading file to */
	char c = ' ';
	const int bufferSize = 10000;
	char buffer[bufferSize];
	char agentname[1000];

	/* Pointer to x-memory for initial state data */
	/*xmachine * current_xmachine;*/
	/* Variables for checking tags */
	int reading, i;
	int in_tag, in_itno, in_xagent, in_name, in_comment;
    int in_vertice_id;
    int in_vertice_value;
    int in_vertice_add_value;
    int in_vertice_previous;
    int in_vertice_max_lag;
    int in_vertice_current_lag;
    int in_vertice_edges;
    int in_vertice_min;
    int in_vertice_max;
    int in_vertice_need_test;
    int in_vertice_correct;
    
    /* tags for environment global variables */
    int in_env;
    int in_env_VERTICES_COUNT;
    
	/* set agent count to zero */
	*h_xmachine_memory_vertice_count = 0;
	
	/* Variables for initial state data */
	int vertice_id;
	float vertice_value;
	float vertice_add_value;
	float vertice_previous;
	int vertice_max_lag;
	int vertice_current_lag;
    float vertice_edges[17];
	float vertice_min;
	float vertice_max;
	int vertice_need_test;
	int vertice_correct;

    /* Variables for environment variables */
    int env_VERTICES_COUNT;
    


	/* Initialise variables */
    initEnvVars();
    agent_maximum.x = 0;
    agent_maximum.y = 0;
    agent_maximum.z = 0;
    agent_minimum.x = 0;
    agent_minimum.y = 0;
    agent_minimum.z = 0;
	reading = 1;
    in_comment = 0;
	in_tag = 0;
	in_itno = 0;
    in_env = 0;
    in_xagent = 0;
	in_name = 0;
	in_vertice_id = 0;
	in_vertice_value = 0;
	in_vertice_add_value = 0;
	in_vertice_previous = 0;
	in_vertice_max_lag = 0;
	in_vertice_current_lag = 0;
	in_vertice_edges = 0;
	in_vertice_min = 0;
	in_vertice_max = 0;
	in_vertice_need_test = 0;
	in_vertice_correct = 0;
    in_env_VERTICES_COUNT = 0;
	//set all vertice values to 0
	//If this is not done then it will cause errors in emu mode where undefined memory is not 0
	for (int k=0; k<xmachine_memory_vertice_MAX; k++)
	{	
		h_vertices->id[k] = 0;
		h_vertices->value[k] = 0;
		h_vertices->add_value[k] = 0;
		h_vertices->previous[k] = 0;
		h_vertices->max_lag[k] = 0;
		h_vertices->current_lag[k] = 0;
        for (i=0;i<17;i++){
            h_vertices->edges[(i*xmachine_memory_vertice_MAX)+k] = 0;
        }
		h_vertices->min[k] = 0;
		h_vertices->max[k] = 0;
		h_vertices->need_test[k] = 0;
		h_vertices->correct[k] = 0;
	}
	

	/* Default variables for memory */
    vertice_id = 0;
    vertice_value = 0;
    vertice_add_value = 0;
    vertice_previous = 0;
    vertice_max_lag = 0;
    vertice_current_lag = 0;
    for (i=0;i<17;i++){
        vertice_edges[i] = 0;
    }
    vertice_min = 0;
    vertice_max = 0;
    vertice_need_test = 0;
    vertice_correct = 0;

    /* Default variables for environment variables */
    env_VERTICES_COUNT = 17;
    


    // Declare and initialise variables tracking the maximum agent id for each agent type from the initial population
    int max_vertice_id = 0;
    
    
    // If no input path was specified, issue a message and return.
    if(inputpath[0] == '\0'){
        printf("No initial states file specified. Using default values.\n");
        return;
    }
    
    // Otherwise an input path was specified, and we have previously checked that it is (was) not a directory. 
    
	// Attempt to open the non directory path as read only.
	file = fopen(inputpath, "r");
    
    // If the file could not be opened, issue a message and return.
    if(file == nullptr)
    {
      printf("Could not open input file %s. Continuing with default values\n", inputpath);
      return;
    }
    // Otherwise we can iterate the file until the end of XML is reached.
    size_t bytesRead = 0;
    i = 0;
	while(reading==1)
	{
        // If I exceeds our buffer size we must abort
        if(i >= bufferSize){
            fprintf(stderr, "Error: XML Parsing failed Tag name or content too long (> %d characters)\n", bufferSize);
            exit(EXIT_FAILURE);
        }

		/* Get the next char from the file */
		c = (char)fgetc(file);

        // Check if we reached the end of the file.
        if(c == EOF){
            // Break out of the loop. This allows for empty files(which may or may not be)
            break;
        }
        // Increment byte counter.
        bytesRead++;

        /*If in a  comment, look for the end of a comment */
        if(in_comment){

            /* Look for an end tag following two (or more) hyphens.
               To support very long comments, we use the minimal amount of buffer we can. 
               If we see a hyphen, store it and increment i (but don't increment i)
               If we see a > check if we have a correct terminating comment
               If we see any other characters, reset i.
            */

            if(c == '-'){
                buffer[i] = c;
                i++;
            } else if(c == '>' && i >= 2){
                in_comment = 0;
                i = 0;
            } else {
                i = 0;
            }

            /*// If we see the end tag, check the preceding two characters for a close comment, if enough characters have been read for -->
            if(c == '>' && i >= 2 && buffer[i-1] == '-' && buffer[i-2] == '-'){
                in_comment = 0;
                buffer[0] = 0;
                i = 0;
            } else {
                // Otherwise just store it in the buffer so we can keep checking for close tags
                buffer[i] = c;
                i++;
            }*/
        }
		/* If the end of a tag */
		else if(c == '>')
		{
			/* Place 0 at end of buffer to make chars a string */
			buffer[i] = 0;

			if(strcmp(buffer, "states") == 0) reading = 1;
			if(strcmp(buffer, "/states") == 0) reading = 0;
			if(strcmp(buffer, "itno") == 0) in_itno = 1;
			if(strcmp(buffer, "/itno") == 0) in_itno = 0;
            if(strcmp(buffer, "environment") == 0) in_env = 1;
            if(strcmp(buffer, "/environment") == 0) in_env = 0;
			if(strcmp(buffer, "name") == 0) in_name = 1;
			if(strcmp(buffer, "/name") == 0) in_name = 0;
            if(strcmp(buffer, "xagent") == 0) in_xagent = 1;
			if(strcmp(buffer, "/xagent") == 0)
			{
				if(strcmp(agentname, "vertice") == 0)
				{
					if (*h_xmachine_memory_vertice_count > xmachine_memory_vertice_MAX){
						printf("ERROR: MAX Buffer size (%i) for agent vertice exceeded whilst reading data\n", xmachine_memory_vertice_MAX);
						// Close the file and stop reading
						fclose(file);
						exit(EXIT_FAILURE);
					}
                    
					h_vertices->id[*h_xmachine_memory_vertice_count] = vertice_id;
					h_vertices->value[*h_xmachine_memory_vertice_count] = vertice_value;
					h_vertices->add_value[*h_xmachine_memory_vertice_count] = vertice_add_value;
					h_vertices->previous[*h_xmachine_memory_vertice_count] = vertice_previous;
					h_vertices->max_lag[*h_xmachine_memory_vertice_count] = vertice_max_lag;
					h_vertices->current_lag[*h_xmachine_memory_vertice_count] = vertice_current_lag;
                    for (int k=0;k<17;k++){
                        h_vertices->edges[(k*xmachine_memory_vertice_MAX)+(*h_xmachine_memory_vertice_count)] = vertice_edges[k];
                    }
					h_vertices->min[*h_xmachine_memory_vertice_count] = vertice_min;
					h_vertices->max[*h_xmachine_memory_vertice_count] = vertice_max;
					h_vertices->need_test[*h_xmachine_memory_vertice_count] = vertice_need_test;
					h_vertices->correct[*h_xmachine_memory_vertice_count] = vertice_correct;
					(*h_xmachine_memory_vertice_count) ++;	
				}
				else
				{
					printf("Warning: agent name undefined - '%s'\n", agentname);
				}



				/* Reset xagent variables */
                vertice_id = 0;
                vertice_value = 0;
                vertice_add_value = 0;
                vertice_previous = 0;
                vertice_max_lag = 0;
                vertice_current_lag = 0;
                for (i=0;i<17;i++){
                    vertice_edges[i] = 0;
                }
                vertice_min = 0;
                vertice_max = 0;
                vertice_need_test = 0;
                vertice_correct = 0;
                
                in_xagent = 0;
			}
			if(strcmp(buffer, "id") == 0) in_vertice_id = 1;
			if(strcmp(buffer, "/id") == 0) in_vertice_id = 0;
			if(strcmp(buffer, "value") == 0) in_vertice_value = 1;
			if(strcmp(buffer, "/value") == 0) in_vertice_value = 0;
			if(strcmp(buffer, "add_value") == 0) in_vertice_add_value = 1;
			if(strcmp(buffer, "/add_value") == 0) in_vertice_add_value = 0;
			if(strcmp(buffer, "previous") == 0) in_vertice_previous = 1;
			if(strcmp(buffer, "/previous") == 0) in_vertice_previous = 0;
			if(strcmp(buffer, "max_lag") == 0) in_vertice_max_lag = 1;
			if(strcmp(buffer, "/max_lag") == 0) in_vertice_max_lag = 0;
			if(strcmp(buffer, "current_lag") == 0) in_vertice_current_lag = 1;
			if(strcmp(buffer, "/current_lag") == 0) in_vertice_current_lag = 0;
			if(strcmp(buffer, "edges") == 0) in_vertice_edges = 1;
			if(strcmp(buffer, "/edges") == 0) in_vertice_edges = 0;
			if(strcmp(buffer, "min") == 0) in_vertice_min = 1;
			if(strcmp(buffer, "/min") == 0) in_vertice_min = 0;
			if(strcmp(buffer, "max") == 0) in_vertice_max = 1;
			if(strcmp(buffer, "/max") == 0) in_vertice_max = 0;
			if(strcmp(buffer, "need_test") == 0) in_vertice_need_test = 1;
			if(strcmp(buffer, "/need_test") == 0) in_vertice_need_test = 0;
			if(strcmp(buffer, "correct") == 0) in_vertice_correct = 1;
			if(strcmp(buffer, "/correct") == 0) in_vertice_correct = 0;
			
            /* environment variables */
            if(strcmp(buffer, "VERTICES_COUNT") == 0) in_env_VERTICES_COUNT = 1;
            if(strcmp(buffer, "/VERTICES_COUNT") == 0) in_env_VERTICES_COUNT = 0;
			

			/* End of tag and reset buffer */
			in_tag = 0;
			i = 0;
		}
		/* If start of tag */
		else if(c == '<')
		{
			/* Place /0 at end of buffer to end numbers */
			buffer[i] = 0;
			/* Flag in tag */
			in_tag = 1;

			if(in_itno) *itno = atoi(buffer);
			if(in_name) strcpy(agentname, buffer);
			else if (in_xagent)
			{
				if(in_vertice_id){
                    vertice_id = (int) fpgu_strtol(buffer); 
                    if(vertice_id > max_vertice_id){
                        max_vertice_id = vertice_id;
                    }
                    
                }
				if(in_vertice_value){
                    vertice_value = (float) fgpu_atof(buffer); 
                }
				if(in_vertice_add_value){
                    vertice_add_value = (float) fgpu_atof(buffer); 
                }
				if(in_vertice_previous){
                    vertice_previous = (float) fgpu_atof(buffer); 
                }
				if(in_vertice_max_lag){
                    vertice_max_lag = (int) fpgu_strtol(buffer); 
                }
				if(in_vertice_current_lag){
                    vertice_current_lag = (int) fpgu_strtol(buffer); 
                }
				if(in_vertice_edges){
                    readArrayInput<float>(&fgpu_atof, buffer, vertice_edges, 17, "vertice", "edges");    
                }
				if(in_vertice_min){
                    vertice_min = (float) fgpu_atof(buffer); 
                }
				if(in_vertice_max){
                    vertice_max = (float) fgpu_atof(buffer); 
                }
				if(in_vertice_need_test){
                    vertice_need_test = (int) fpgu_strtol(buffer); 
                }
				if(in_vertice_correct){
                    vertice_correct = (int) fpgu_strtol(buffer); 
                }
				
            }
            else if (in_env){
            if(in_env_VERTICES_COUNT){
              
                    env_VERTICES_COUNT = (int) fpgu_strtol(buffer);
                    
                    set_VERTICES_COUNT(&env_VERTICES_COUNT);
                  
              }
            
            }
		/* Reset buffer */
			i = 0;
		}
		/* If in tag put read char into buffer */
		else if(in_tag)
		{
            // Check if we are a comment, when we are in a tag and buffer[0:2] == "!--"
            if(i == 2 && c == '-' && buffer[1] == '-' && buffer[0] == '!'){
                in_comment = 1;
                // Reset the buffer and i.
                buffer[0] = 0;
                i = 0;
            }

            // Store the character and increment the counter
            buffer[i] = c;
            i++;

		}
		/* If in data read char into buffer */
		else
		{
			buffer[i] = c;
			i++;
		}
	}
    // If no bytes were read, raise a warning.
    if(bytesRead == 0){
        fprintf(stdout, "Warning: %s is an empty file\n", inputpath);
        fflush(stdout);
    }

    // If the in_comment flag is still marked, issue a warning.
    if(in_comment){
        fprintf(stdout, "Warning: Un-terminated comment in %s\n", inputpath);
        fflush(stdout);
    }    

	/* Close the file */
	fclose(file);

    // IF required, set the first id value to maximum plus one.
    
    // If any agents of this type were found, use the maximum value +1
    if(h_xmachine_memory_vertice_count > 0){
        set_initial_vertice_id(max_vertice_id + 1);

    } else {
    // Otherwise use 0.
        set_initial_vertice_id(0);
    }


    
}

glm::vec3 getMaximumBounds(){
    return agent_maximum;
}

glm::vec3 getMinimumBounds(){
    return agent_minimum;
}


/* Methods to load static networks from disk */
