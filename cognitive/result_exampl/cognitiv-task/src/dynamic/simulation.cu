#include "hip/hip_runtime.h"

/*
 * FLAME GPU v 1.5.X for CUDA 9
 * Copyright University of Sheffield.
 * Original Author: Dr Paul Richmond (user contributions tracked on https://github.com/FLAMEGPU/FLAMEGPU)
 * Contact: p.richmond@sheffield.ac.uk (http://www.paulrichmond.staff.shef.ac.uk)
 *
 * University of Sheffield retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * University of Sheffield is strictly prohibited.
 *
 * For terms of licence agreement please attached licence or view licence
 * on www.flamegpu.com website.
 *
 */


  //Disable internal thrust warnings about conversions
  #ifdef _MSC_VER
  #pragma warning(push)
  #pragma warning (disable : 4267)
  #pragma warning (disable : 4244)
  #endif
  #ifdef __GNUC__
  #pragma GCC diagnostic push
  #pragma GCC diagnostic ignored "-Wunused-parameter"
  #endif

  // includes
  #include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <cmath>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/system/cuda/execution_policy.h>
#include <hipcub/hipcub.hpp>

// include FLAME kernels
#include "FLAMEGPU_kernals.cu"


#ifdef _MSC_VER
#pragma warning(pop)
#endif
#ifdef __GNUC__
#pragma GCC diagnostic pop
#endif

/* Error check function for safe CUDA API calling */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Error check function for post CUDA Kernel calling */
#define gpuErrchkLaunch() { gpuLaunchAssert(__FILE__, __LINE__); }
inline void gpuLaunchAssert(const char *file, int line, bool abort=true)
{
	gpuAssert( hipPeekAtLastError(), file, line );
#ifdef _DEBUG
	gpuAssert( hipDeviceSynchronize(), file, line );
#endif
   
}

/* SM padding and offset variables */
int SM_START;
int PADDING;

unsigned int g_iterationNumber;

/* Agent Memory */

/* vertice Agent variables these lists are used in the agent function where as the other lists are used only outside the agent functions*/
xmachine_memory_vertice_list* d_vertices;      /**< Pointer to agent list (population) on the device*/
xmachine_memory_vertice_list* d_vertices_swap; /**< Pointer to agent list swap on the device (used when killing agents)*/
xmachine_memory_vertice_list* d_vertices_new;  /**< Pointer to new agent list on the device (used to hold new agents before they are appended to the population)*/
int h_xmachine_memory_vertice_count;   /**< Agent population size counter */ 
uint * d_xmachine_memory_vertice_keys;	  /**< Agent sort identifiers keys*/
uint * d_xmachine_memory_vertice_values;  /**< Agent sort identifiers value */

/* vertice state variables */
xmachine_memory_vertice_list* h_vertices_default;      /**< Pointer to agent list (population) on host*/
xmachine_memory_vertice_list* d_vertices_default;      /**< Pointer to agent list (population) on the device*/
int h_xmachine_memory_vertice_default_count;   /**< Agent population size counter */ 


/* Variables to track the state of host copies of state lists, for the purposes of host agent data access.
 * @future - if the host data is current it may be possible to avoid duplicating memcpy in xml output.
 */
unsigned int h_vertices_default_variable_id_data_iteration;
unsigned int h_vertices_default_variable_value_data_iteration;
unsigned int h_vertices_default_variable_add_value_data_iteration;
unsigned int h_vertices_default_variable_previous_data_iteration;
unsigned int h_vertices_default_variable_max_lag_data_iteration;
unsigned int h_vertices_default_variable_current_lag_data_iteration;
unsigned int h_vertices_default_variable_edges_data_iteration;
unsigned int h_vertices_default_variable_min_data_iteration;
unsigned int h_vertices_default_variable_max_data_iteration;
unsigned int h_vertices_default_variable_need_test_data_iteration;
unsigned int h_vertices_default_variable_correct_data_iteration;


/* Message Memory */

/* send_local Message variables */
xmachine_message_send_local_list* h_send_locals;         /**< Pointer to message list on host*/
xmachine_message_send_local_list* d_send_locals;         /**< Pointer to message list on device*/
xmachine_message_send_local_list* d_send_locals_swap;    /**< Pointer to message swap list on device (used for holding optional messages)*/
/* Non partitioned and spatial partitioned message variables  */
int h_message_send_local_count;         /**< message list counter*/
int h_message_send_local_output_type;   /**< message output type (single or optional)*/

  
/* CUDA Streams for function layers */
hipStream_t stream1;

/* Device memory and sizes for CUB values */

void * d_temp_scan_storage_vertice;
size_t temp_scan_storage_bytes_vertice;


/*Global condition counts*/

/* Agent ID Generation functions implemented in simulation.cu and FLAMEGPU_kernals.cu*/
int h_current_value_generate_vertice_id = 0;

// Track the last value returned from the device, to enable copying to the device after a step function.
int h_last_value_generate_vertice_id = INT_MAX;

void set_initial_vertice_id(int firstID){
  h_current_value_generate_vertice_id = firstID;
}

// Function to copy from the host to the device in the default stream
void update_device_generate_vertice_id(){
// If the last device value doesn't match the current value, update the device value. 
  if(h_current_value_generate_vertice_id != h_last_value_generate_vertice_id){
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_current_value_generate_vertice_id), &h_current_value_generate_vertice_id, sizeof(int)));
  }
}
// Function to copy from the device to the host in the default stream
void update_host_generate_vertice_id(){
  gpuErrchk(hipMemcpyFromSymbol( &h_current_value_generate_vertice_id, HIP_SYMBOL(d_current_value_generate_vertice_id), sizeof(int)));
  h_last_value_generate_vertice_id = h_current_value_generate_vertice_id;
}



/* RNG rand48 */
RNG_rand48* h_rand48;    /**< Pointer to RNG_rand48 seed list on host*/
RNG_rand48* d_rand48;    /**< Pointer to RNG_rand48 seed list on device*/

/* Early simulation exit*/
bool g_exit_early;

/* Cuda Event Timers for Instrumentation */
#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEvent_t instrument_iteration_start, instrument_iteration_stop;
	float instrument_iteration_milliseconds = 0.0f;
#endif
#if (defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS) || (defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS) || (defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS) || (defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS)
	hipEvent_t instrument_start, instrument_stop;
	float instrument_milliseconds = 0.0f;
#endif

/* CUDA Parallel Primatives variables */
int scan_last_sum;           /**< Indicates if the position (in message list) of last message*/
int scan_last_included;      /**< Indicates if last sum value is included in the total sum count*/

/* Agent function prototypes */

/** vertice_send_message
 * Agent function prototype for send_message function of vertice agent
 */
void vertice_send_message(hipStream_t &stream);

/** vertice_read_message
 * Agent function prototype for read_message function of vertice agent
 */
void vertice_read_message(hipStream_t &stream);

  
void setPaddingAndOffset()
{
    PROFILE_SCOPED_RANGE("setPaddingAndOffset");
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	int x64_sys = 0;

	// This function call returns 9999 for both major & minor fields, if no CUDA capable devices are present
	if (deviceProp.major == 9999 && deviceProp.minor == 9999){
		printf("Error: There is no device supporting CUDA.\n");
		exit(EXIT_FAILURE);
	}
    
    //check if double is used and supported
#ifdef _DOUBLE_SUPPORT_REQUIRED_
	printf("Simulation requires full precision double values\n");
	if ((deviceProp.major < 2)&&(deviceProp.minor < 3)){
		printf("Error: Hardware does not support full precision double values!\n");
		exit(EXIT_FAILURE);
	}
    
#endif

	//check 32 or 64bit
	x64_sys = (sizeof(void*)==8);
	if (x64_sys)
	{
		printf("64Bit System Detected\n");
	}
	else
	{
		printf("32Bit System Detected\n");
	}

	SM_START = 0;
	PADDING = 0;
  
	//copy padding and offset to GPU
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_SM_START), &SM_START, sizeof(int)));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_PADDING), &PADDING, sizeof(int)));     
}

int is_sqr_pow2(int x){
	int r = (int)pow(4, ceil(log(x)/log(4)));
	return (r == x);
}

int lowest_sqr_pow2(int x){
	int l;
	
	//escape early if x is square power of 2
	if (is_sqr_pow2(x))
		return x;
	
	//lower bound		
	l = (int)pow(4, floor(log(x)/log(4)));
	
	return l;
}

/* Unary function required for hipOccupancyMaxPotentialBlockSizeVariableSMem to avoid warnings */
int no_sm(int b){
	return 0;
}

/* Unary function to return shared memory size for reorder message kernels */
int reorder_messages_sm_size(int blockSize)
{
	return sizeof(unsigned int)*(blockSize+1);
}


/** getIterationNumber
 *  Get the iteration number (host)
 *  @return a 1 indexed value for the iteration number, which is incremented at the start of each simulation step.
 *      I.e. it is 0 on up until the first call to singleIteration()
 */
extern unsigned int getIterationNumber(){
    return g_iterationNumber;
}

void initialise(char * inputfile){
    PROFILE_SCOPED_RANGE("initialise");

	//set the padding and offset values depending on architecture and OS
	setPaddingAndOffset();
  
		// Initialise some global variables
		g_iterationNumber = 0;
		g_exit_early = false;

    // Initialise variables for tracking which iterations' data is accessible on the host.
    h_vertices_default_variable_id_data_iteration = 0;
    h_vertices_default_variable_value_data_iteration = 0;
    h_vertices_default_variable_add_value_data_iteration = 0;
    h_vertices_default_variable_previous_data_iteration = 0;
    h_vertices_default_variable_max_lag_data_iteration = 0;
    h_vertices_default_variable_current_lag_data_iteration = 0;
    h_vertices_default_variable_edges_data_iteration = 0;
    h_vertices_default_variable_min_data_iteration = 0;
    h_vertices_default_variable_max_data_iteration = 0;
    h_vertices_default_variable_need_test_data_iteration = 0;
    h_vertices_default_variable_correct_data_iteration = 0;
    



	printf("Allocating Host and Device memory\n");
    PROFILE_PUSH_RANGE("allocate host");
	/* Agent memory allocation (CPU) */
	int xmachine_vertice_SoA_size = sizeof(xmachine_memory_vertice_list);
	h_vertices_default = (xmachine_memory_vertice_list*)malloc(xmachine_vertice_SoA_size);

	/* Message memory allocation (CPU) */
	int message_send_local_SoA_size = sizeof(xmachine_message_send_local_list);
	h_send_locals = (xmachine_message_send_local_list*)malloc(message_send_local_SoA_size);

	//Exit if agent or message buffer sizes are to small for function outputs

  /* Graph memory allocation (CPU) */
  

    PROFILE_POP_RANGE(); //"allocate host"
	

	//read initial states
	readInitialStates(inputfile, h_vertices_default, &h_xmachine_memory_vertice_default_count);

  // Read graphs from disk
  

  PROFILE_PUSH_RANGE("allocate device");
	
	/* vertice Agent memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_vertices, xmachine_vertice_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_vertices_swap, xmachine_vertice_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_vertices_new, xmachine_vertice_SoA_size));
    //continuous agent sort identifiers
  gpuErrchk( hipMalloc( (void**) &d_xmachine_memory_vertice_keys, xmachine_memory_vertice_MAX* sizeof(uint)));
	gpuErrchk( hipMalloc( (void**) &d_xmachine_memory_vertice_values, xmachine_memory_vertice_MAX* sizeof(uint)));
	/* default memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_vertices_default, xmachine_vertice_SoA_size));
	gpuErrchk( hipMemcpy( d_vertices_default, h_vertices_default, xmachine_vertice_SoA_size, hipMemcpyHostToDevice));
    
	/* send_local Message memory allocation (GPU) */
	gpuErrchk( hipMalloc( (void**) &d_send_locals, message_send_local_SoA_size));
	gpuErrchk( hipMalloc( (void**) &d_send_locals_swap, message_send_local_SoA_size));
	gpuErrchk( hipMemcpy( d_send_locals, h_send_locals, message_send_local_SoA_size, hipMemcpyHostToDevice));
		


  /* Allocate device memory for graphs */
  

    PROFILE_POP_RANGE(); // "allocate device"

    /* Calculate and allocate CUB temporary memory for exclusive scans */
    
    d_temp_scan_storage_vertice = nullptr;
    temp_scan_storage_bytes_vertice = 0;
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_scan_storage_vertice, 
        temp_scan_storage_bytes_vertice, 
        (int*) nullptr, 
        (int*) nullptr, 
        xmachine_memory_vertice_MAX
    );
    gpuErrchk(hipMalloc(&d_temp_scan_storage_vertice, temp_scan_storage_bytes_vertice));
    

	/*Set global condition counts*/

	/* RNG rand48 */
    PROFILE_PUSH_RANGE("Initialse RNG_rand48");
	int h_rand48_SoA_size = sizeof(RNG_rand48);
	h_rand48 = (RNG_rand48*)malloc(h_rand48_SoA_size);
	//allocate on GPU
	gpuErrchk( hipMalloc( (void**) &d_rand48, h_rand48_SoA_size));
	// calculate strided iteration constants
	static const unsigned long long a = 0x5DEECE66DLL, c = 0xB;
	int seed = 123;
	unsigned long long A, C;
	A = 1LL; C = 0LL;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		C += A*c;
		A *= a;
	}
	h_rand48->A.x = A & 0xFFFFFFLL;
	h_rand48->A.y = (A >> 24) & 0xFFFFFFLL;
	h_rand48->C.x = C & 0xFFFFFFLL;
	h_rand48->C.y = (C >> 24) & 0xFFFFFFLL;
	// prepare first nThreads random numbers from seed
	unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
	for (unsigned int i = 0; i < buffer_size_MAX; ++i) {
		x = a*x + c;
		h_rand48->seeds[i].x = x & 0xFFFFFFLL;
		h_rand48->seeds[i].y = (x >> 24) & 0xFFFFFFLL;
	}
	//copy to device
	gpuErrchk( hipMemcpy( d_rand48, h_rand48, h_rand48_SoA_size, hipMemcpyHostToDevice));

    PROFILE_POP_RANGE();

	/* Call all init functions */
	/* Prepare cuda event timers for instrumentation */
#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventCreate(&instrument_iteration_start);
	hipEventCreate(&instrument_iteration_stop);
#endif
#if (defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS) || (defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS) || (defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS) || (defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS)
	hipEventCreate(&instrument_start);
	hipEventCreate(&instrument_stop);
#endif

	

  /* If any Agents can generate IDs, update the device value after init functions have executed */

  update_device_generate_vertice_id();

  
  /* Init CUDA Streams for function layers */
  
  gpuErrchk(hipStreamCreate(&stream1));

#if defined(OUTPUT_POPULATION_PER_ITERATION) && OUTPUT_POPULATION_PER_ITERATION
	// Print the agent population size of all agents in all states
	
		printf("Init agent_vertice_default_count: %u\n",get_agent_vertice_default_count());
	
#endif
} 


void sort_vertices_default(void (*generate_key_value_pairs)(unsigned int* keys, unsigned int* values, xmachine_memory_vertice_list* agents))
{
	int blockSize;
	int minGridSize;
	int gridSize;

	//generate sort keys
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, generate_key_value_pairs, no_sm, h_xmachine_memory_vertice_default_count); 
	gridSize = (h_xmachine_memory_vertice_default_count + blockSize - 1) / blockSize;    // Round up according to array size 
	generate_key_value_pairs<<<gridSize, blockSize>>>(d_xmachine_memory_vertice_keys, d_xmachine_memory_vertice_values, d_vertices_default);
	gpuErrchkLaunch();

	//updated Thrust sort
	thrust::sort_by_key( thrust::device_pointer_cast(d_xmachine_memory_vertice_keys),  thrust::device_pointer_cast(d_xmachine_memory_vertice_keys) + h_xmachine_memory_vertice_default_count,  thrust::device_pointer_cast(d_xmachine_memory_vertice_values));
	gpuErrchkLaunch();

	//reorder agents
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reorder_vertice_agents, no_sm, h_xmachine_memory_vertice_default_count); 
	gridSize = (h_xmachine_memory_vertice_default_count + blockSize - 1) / blockSize;    // Round up according to array size 
	reorder_vertice_agents<<<gridSize, blockSize>>>(d_xmachine_memory_vertice_values, d_vertices_default, d_vertices_swap);
	gpuErrchkLaunch();

	//swap
	xmachine_memory_vertice_list* d_vertices_temp = d_vertices_default;
	d_vertices_default = d_vertices_swap;
	d_vertices_swap = d_vertices_temp;	
}


void cleanup(){
    PROFILE_SCOPED_RANGE("cleanup");

    /* Call all exit functions */
	

	/* Agent data free*/
	
	/* vertice Agent variables */
	gpuErrchk(hipFree(d_vertices));
	gpuErrchk(hipFree(d_vertices_swap));
	gpuErrchk(hipFree(d_vertices_new));
	
	free( h_vertices_default);
	gpuErrchk(hipFree(d_vertices_default));
	

	/* Message data free */
	
	/* send_local Message variables */
	free( h_send_locals);
	gpuErrchk(hipFree(d_send_locals));
	gpuErrchk(hipFree(d_send_locals_swap));
	

    /* Free temporary CUB memory if required. */
    
    if(d_temp_scan_storage_vertice != nullptr){
      gpuErrchk(hipFree(d_temp_scan_storage_vertice));
      d_temp_scan_storage_vertice = nullptr;
      temp_scan_storage_bytes_vertice = 0;
    }
    

  /* Graph data free */
  
  
  /* CUDA Streams for function layers */
  
  gpuErrchk(hipStreamDestroy(stream1));

  /* CUDA Event Timers for Instrumentation */
#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventDestroy(instrument_iteration_start);
	hipEventDestroy(instrument_iteration_stop);
#endif
#if (defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS) || (defined(INSTRUMENT_INIT_FUNCTIONS) && INSTRUMENT_INIT_FUNCTIONS) || (defined(INSTRUMENT_STEP_FUNCTIONS) && INSTRUMENT_STEP_FUNCTIONS) || (defined(INSTRUMENT_EXIT_FUNCTIONS) && INSTRUMENT_EXIT_FUNCTIONS)
	hipEventDestroy(instrument_start);
	hipEventDestroy(instrument_stop);
#endif
}

void singleIteration(){
PROFILE_SCOPED_RANGE("singleIteration");

#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventRecord(instrument_iteration_start);
#endif

    // Increment the iteration number.
    g_iterationNumber++;

  /* set all non partitioned, spatial partitioned and On-Graph Partitioned message counts to 0*/
	h_message_send_local_count = 0;
	//upload to device constant
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL( d_message_send_local_count), &h_message_send_local_count, sizeof(int)));
	

	/* Call agent functions in order iterating through the layer functions */
	
	/* Layer 1*/
	
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
	
    PROFILE_PUSH_RANGE("vertice_send_message");
	vertice_send_message(stream1);
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: vertice_send_message = %f (ms)\n", instrument_milliseconds);
#endif
	hipDeviceSynchronize();
  
	/* Layer 2*/
	
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_start);
#endif
	
    PROFILE_PUSH_RANGE("vertice_read_message");
	vertice_read_message(stream1);
    PROFILE_POP_RANGE();
#if defined(INSTRUMENT_AGENT_FUNCTIONS) && INSTRUMENT_AGENT_FUNCTIONS
	hipEventRecord(instrument_stop);
	hipEventSynchronize(instrument_stop);
	hipEventElapsedTime(&instrument_milliseconds, instrument_start, instrument_stop);
	printf("Instrumentation: vertice_read_message = %f (ms)\n", instrument_milliseconds);
#endif
	hipDeviceSynchronize();
  

  /* If any Agents can generate IDs, update the host value after agent functions have executed */

  update_host_generate_vertice_id();

    
    /* Call all step functions */
	

/* If any Agents can generate IDs, update the device value after step functions have executed */

  update_device_generate_vertice_id();


#if defined(OUTPUT_POPULATION_PER_ITERATION) && OUTPUT_POPULATION_PER_ITERATION
	// Print the agent population size of all agents in all states
	
		printf("agent_vertice_default_count: %u\n",get_agent_vertice_default_count());
	
#endif

#if defined(INSTRUMENT_ITERATIONS) && INSTRUMENT_ITERATIONS
	hipEventRecord(instrument_iteration_stop);
	hipEventSynchronize(instrument_iteration_stop);
	hipEventElapsedTime(&instrument_iteration_milliseconds, instrument_iteration_start, instrument_iteration_stop);
	printf("Instrumentation: Iteration Time = %f (ms)\n", instrument_iteration_milliseconds);
#endif
}

/* finish whole simulation after this step */
void set_exit_early() {
	g_exit_early = true;
}

bool get_exit_early() {
	return g_exit_early;
}

/* Environment functions */

//host constant declaration
int h_env_VERTICES_COUNT;


//constant setter
void set_VERTICES_COUNT(int* h_VERTICES_COUNT){
    gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(VERTICES_COUNT), h_VERTICES_COUNT, sizeof(int)));
    memcpy(&h_env_VERTICES_COUNT, h_VERTICES_COUNT,sizeof(int));
}

//constant getter
const int* get_VERTICES_COUNT(){
    return &h_env_VERTICES_COUNT;
}




/* Agent data access functions*/

    
int get_agent_vertice_MAX_count(){
    return xmachine_memory_vertice_MAX;
}


int get_agent_vertice_default_count(){
	//continuous agent
	return h_xmachine_memory_vertice_default_count;
	
}

xmachine_memory_vertice_list* get_device_vertice_default_agents(){
	return d_vertices_default;
}

xmachine_memory_vertice_list* get_host_vertice_default_agents(){
	return h_vertices_default;
}



/* Host based access of agent variables*/

/** int get_vertice_default_variable_id(unsigned int index)
 * Gets the value of the id variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable id
 */
__host__ int get_vertice_default_variable_id(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_id_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->id,
                    d_vertices_default->id,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_id_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->id[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access id for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** float get_vertice_default_variable_value(unsigned int index)
 * Gets the value of the value variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable value
 */
__host__ float get_vertice_default_variable_value(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_value_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->value,
                    d_vertices_default->value,
                    count * sizeof(float),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_value_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->value[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access value for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** float get_vertice_default_variable_add_value(unsigned int index)
 * Gets the value of the add_value variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable add_value
 */
__host__ float get_vertice_default_variable_add_value(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_add_value_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->add_value,
                    d_vertices_default->add_value,
                    count * sizeof(float),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_add_value_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->add_value[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access add_value for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** float get_vertice_default_variable_previous(unsigned int index)
 * Gets the value of the previous variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable previous
 */
__host__ float get_vertice_default_variable_previous(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_previous_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->previous,
                    d_vertices_default->previous,
                    count * sizeof(float),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_previous_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->previous[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access previous for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** int get_vertice_default_variable_max_lag(unsigned int index)
 * Gets the value of the max_lag variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable max_lag
 */
__host__ int get_vertice_default_variable_max_lag(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_max_lag_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->max_lag,
                    d_vertices_default->max_lag,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_max_lag_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->max_lag[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access max_lag for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** int get_vertice_default_variable_current_lag(unsigned int index)
 * Gets the value of the current_lag variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable current_lag
 */
__host__ int get_vertice_default_variable_current_lag(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_current_lag_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->current_lag,
                    d_vertices_default->current_lag,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_current_lag_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->current_lag[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access current_lag for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** float get_vertice_default_variable_edges(unsigned int index, unsigned int element)
 * Gets the element-th value of the edges variable array of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @param element the element index within the variable array
 * @return element-th value of agent variable edges
 */
__host__ float get_vertice_default_variable_edges(unsigned int index, unsigned int element){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int numElements = 17;
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count && element < numElements ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_edges_data_iteration != currentIteration){
            
            for(unsigned int e = 0; e < numElements; e++){
                gpuErrchk(
                    hipMemcpy(
                        h_vertices_default->edges + (e * xmachine_memory_vertice_MAX),
                        d_vertices_default->edges + (e * xmachine_memory_vertice_MAX), 
                        count * sizeof(float), 
                        hipMemcpyDeviceToHost
                    )
                );
                // Update some global value indicating what data is currently present in that host array.
                h_vertices_default_variable_edges_data_iteration = currentIteration;
            }
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->edges[index + (element * xmachine_memory_vertice_MAX)];

    } else {
        fprintf(stderr, "Warning: Attempting to access the %u-th element of edges for the %u th member of vertice_default. count is %u at iteration %u\n", element, index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** float get_vertice_default_variable_min(unsigned int index)
 * Gets the value of the min variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable min
 */
__host__ float get_vertice_default_variable_min(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_min_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->min,
                    d_vertices_default->min,
                    count * sizeof(float),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_min_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->min[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access min for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** float get_vertice_default_variable_max(unsigned int index)
 * Gets the value of the max variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable max
 */
__host__ float get_vertice_default_variable_max(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_max_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->max,
                    d_vertices_default->max,
                    count * sizeof(float),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_max_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->max[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access max for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** int get_vertice_default_variable_need_test(unsigned int index)
 * Gets the value of the need_test variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable need_test
 */
__host__ int get_vertice_default_variable_need_test(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_need_test_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->need_test,
                    d_vertices_default->need_test,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_need_test_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->need_test[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access need_test for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}

/** int get_vertice_default_variable_correct(unsigned int index)
 * Gets the value of the correct variable of an vertice agent in the default state on the host. 
 * If the data is not currently on the host, a memcpy of the data of all agents in that state list will be issued, via a global.
 * This has a potentially significant performance impact if used improperly.
 * @param index the index of the agent within the list.
 * @return value of agent variable correct
 */
__host__ int get_vertice_default_variable_correct(unsigned int index){
    unsigned int count = get_agent_vertice_default_count();
    unsigned int currentIteration = getIterationNumber();
    
    // If the index is within bounds - no need to check >= 0 due to unsigned.
    if(count > 0 && index < count ){
        // If necessary, copy agent data from the device to the host in the default stream
        if(h_vertices_default_variable_correct_data_iteration != currentIteration){
            gpuErrchk(
                hipMemcpy(
                    h_vertices_default->correct,
                    d_vertices_default->correct,
                    count * sizeof(int),
                    hipMemcpyDeviceToHost
                )
            );
            // Update some global value indicating what data is currently present in that host array.
            h_vertices_default_variable_correct_data_iteration = currentIteration;
        }

        // Return the value of the index-th element of the relevant host array.
        return h_vertices_default->correct[index];

    } else {
        fprintf(stderr, "Warning: Attempting to access correct for the %u th member of vertice_default. count is %u at iteration %u\n", index, count, currentIteration);
        // Otherwise we return a default value
        return 0;

    }
}



/* Host based agent creation functions */
// These are only available for continuous agents.



/* copy_single_xmachine_memory_vertice_hostToDevice
 * Private function to copy a host agent struct into a device SoA agent list.
 * @param d_dst destination agent state list
 * @param h_agent agent struct
 */
void copy_single_xmachine_memory_vertice_hostToDevice(xmachine_memory_vertice_list * d_dst, xmachine_memory_vertice * h_agent){
 
		gpuErrchk(hipMemcpy(d_dst->id, &h_agent->id, sizeof(int), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->value, &h_agent->value, sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->add_value, &h_agent->add_value, sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->previous, &h_agent->previous, sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->max_lag, &h_agent->max_lag, sizeof(int), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->current_lag, &h_agent->current_lag, sizeof(int), hipMemcpyHostToDevice));
 
	for(unsigned int i = 0; i < 17; i++){
		gpuErrchk(hipMemcpy(d_dst->edges + (i * xmachine_memory_vertice_MAX), h_agent->edges + i, sizeof(float), hipMemcpyHostToDevice));
    }
 
		gpuErrchk(hipMemcpy(d_dst->min, &h_agent->min, sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->max, &h_agent->max, sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->need_test, &h_agent->need_test, sizeof(int), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->correct, &h_agent->correct, sizeof(int), hipMemcpyHostToDevice));

}
/*
 * Private function to copy some elements from a host based struct of arrays to a device based struct of arrays for a single agent state.
 * Individual copies of `count` elements are performed for each agent variable or each component of agent array variables, to avoid wasted data transfer.
 * There will be a point at which a single hipMemcpy will outperform many smaller memcpys, however host based agent creation should typically only populate a fraction of the maximum buffer size, so this should be more efficient.
 * @optimisation - experimentally find the proportion at which transferring the whole SoA would be better and incorporate this. The same will apply to agent variable arrays.
 * 
 * @param d_dst device destination SoA
 * @oaram h_src host source SoA
 * @param count the number of agents to transfer data for
 */
void copy_partial_xmachine_memory_vertice_hostToDevice(xmachine_memory_vertice_list * d_dst, xmachine_memory_vertice_list * h_src, unsigned int count){
    // Only copy elements if there is data to move.
    if (count > 0){
	 
		gpuErrchk(hipMemcpy(d_dst->id, h_src->id, count * sizeof(int), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->value, h_src->value, count * sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->add_value, h_src->add_value, count * sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->previous, h_src->previous, count * sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->max_lag, h_src->max_lag, count * sizeof(int), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->current_lag, h_src->current_lag, count * sizeof(int), hipMemcpyHostToDevice));
 
		for(unsigned int i = 0; i < 17; i++){
			gpuErrchk(hipMemcpy(d_dst->edges + (i * xmachine_memory_vertice_MAX), h_src->edges + (i * xmachine_memory_vertice_MAX), count * sizeof(float), hipMemcpyHostToDevice));
        }

 
		gpuErrchk(hipMemcpy(d_dst->min, h_src->min, count * sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->max, h_src->max, count * sizeof(float), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->need_test, h_src->need_test, count * sizeof(int), hipMemcpyHostToDevice));
 
		gpuErrchk(hipMemcpy(d_dst->correct, h_src->correct, count * sizeof(int), hipMemcpyHostToDevice));

    }
}

xmachine_memory_vertice* h_allocate_agent_vertice(){
	xmachine_memory_vertice* agent = (xmachine_memory_vertice*)malloc(sizeof(xmachine_memory_vertice));
	// Memset the whole agent strcuture
    memset(agent, 0, sizeof(xmachine_memory_vertice));

    agent->add_value = 0;

    agent->current_lag = 0;
	// Agent variable arrays must be allocated
    agent->edges = (float*)malloc(17 * sizeof(float));
	
    // If there is no default value, memset to 0.
    memset(agent->edges, 0, sizeof(float)*17);
	return agent;
}
void h_free_agent_vertice(xmachine_memory_vertice** agent){

    free((*agent)->edges);
 
	free((*agent));
	(*agent) = NULL;
}
xmachine_memory_vertice** h_allocate_agent_vertice_array(unsigned int count){
	xmachine_memory_vertice ** agents = (xmachine_memory_vertice**)malloc(count * sizeof(xmachine_memory_vertice*));
	for (unsigned int i = 0; i < count; i++) {
		agents[i] = h_allocate_agent_vertice();
	}
	return agents;
}
void h_free_agent_vertice_array(xmachine_memory_vertice*** agents, unsigned int count){
	for (unsigned int i = 0; i < count; i++) {
		h_free_agent_vertice(&((*agents)[i]));
	}
	free((*agents));
	(*agents) = NULL;
}

void h_unpack_agents_vertice_AoS_to_SoA(xmachine_memory_vertice_list * dst, xmachine_memory_vertice** src, unsigned int count){
	if(count > 0){
		for(unsigned int i = 0; i < count; i++){
			 
			dst->id[i] = src[i]->id;
			 
			dst->value[i] = src[i]->value;
			 
			dst->add_value[i] = src[i]->add_value;
			 
			dst->previous[i] = src[i]->previous;
			 
			dst->max_lag[i] = src[i]->max_lag;
			 
			dst->current_lag[i] = src[i]->current_lag;
			 
			for(unsigned int j = 0; j < 17; j++){
				dst->edges[(j * xmachine_memory_vertice_MAX) + i] = src[i]->edges[j];
			}
			 
			dst->min[i] = src[i]->min;
			 
			dst->max[i] = src[i]->max;
			 
			dst->need_test[i] = src[i]->need_test;
			 
			dst->correct[i] = src[i]->correct;
			
		}
	}
}


void h_add_agent_vertice_default(xmachine_memory_vertice* agent){
	if (h_xmachine_memory_vertice_count + 1 > xmachine_memory_vertice_MAX){
		printf("Error: Buffer size of vertice agents in state default will be exceeded by h_add_agent_vertice_default\n");
		exit(EXIT_FAILURE);
	}	

	int blockSize;
	int minGridSize;
	int gridSize;
	unsigned int count = 1;
	
	// Copy data from host struct to device SoA for target state
	copy_single_xmachine_memory_vertice_hostToDevice(d_vertices_new, agent);

	// Use append kernel (@optimisation - This can be replaced with a pointer swap if the target state list is empty)
	hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, append_vertice_Agents, no_sm, count);
	gridSize = (count + blockSize - 1) / blockSize;
	append_vertice_Agents <<<gridSize, blockSize, 0, stream1 >>>(d_vertices_default, d_vertices_new, h_xmachine_memory_vertice_default_count, count);
	gpuErrchkLaunch();
	// Update the number of agents in this state.
	h_xmachine_memory_vertice_default_count += count;
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_xmachine_memory_vertice_default_count), &h_xmachine_memory_vertice_default_count, sizeof(int)));
	hipDeviceSynchronize();

    // Reset host variable status flags for the relevant agent state list as the device state list has been modified.
    h_vertices_default_variable_id_data_iteration = 0;
    h_vertices_default_variable_value_data_iteration = 0;
    h_vertices_default_variable_add_value_data_iteration = 0;
    h_vertices_default_variable_previous_data_iteration = 0;
    h_vertices_default_variable_max_lag_data_iteration = 0;
    h_vertices_default_variable_current_lag_data_iteration = 0;
    h_vertices_default_variable_edges_data_iteration = 0;
    h_vertices_default_variable_min_data_iteration = 0;
    h_vertices_default_variable_max_data_iteration = 0;
    h_vertices_default_variable_need_test_data_iteration = 0;
    h_vertices_default_variable_correct_data_iteration = 0;
    

}
void h_add_agents_vertice_default(xmachine_memory_vertice** agents, unsigned int count){
	if(count > 0){
		int blockSize;
		int minGridSize;
		int gridSize;

		if (h_xmachine_memory_vertice_count + count > xmachine_memory_vertice_MAX){
			printf("Error: Buffer size of vertice agents in state default will be exceeded by h_add_agents_vertice_default\n");
			exit(EXIT_FAILURE);
		}

		// Unpack data from AoS into the pre-existing SoA
		h_unpack_agents_vertice_AoS_to_SoA(h_vertices_default, agents, count);

		// Copy data from the host SoA to the device SoA for the target state
		copy_partial_xmachine_memory_vertice_hostToDevice(d_vertices_new, h_vertices_default, count);

		// Use append kernel (@optimisation - This can be replaced with a pointer swap if the target state list is empty)
		hipOccupancyMaxPotentialBlockSizeVariableSMem(&minGridSize, &blockSize, append_vertice_Agents, no_sm, count);
		gridSize = (count + blockSize - 1) / blockSize;
		append_vertice_Agents <<<gridSize, blockSize, 0, stream1 >>>(d_vertices_default, d_vertices_new, h_xmachine_memory_vertice_default_count, count);
		gpuErrchkLaunch();
		// Update the number of agents in this state.
		h_xmachine_memory_vertice_default_count += count;
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_xmachine_memory_vertice_default_count), &h_xmachine_memory_vertice_default_count, sizeof(int)));
		hipDeviceSynchronize();

        // Reset host variable status flags for the relevant agent state list as the device state list has been modified.
        h_vertices_default_variable_id_data_iteration = 0;
        h_vertices_default_variable_value_data_iteration = 0;
        h_vertices_default_variable_add_value_data_iteration = 0;
        h_vertices_default_variable_previous_data_iteration = 0;
        h_vertices_default_variable_max_lag_data_iteration = 0;
        h_vertices_default_variable_current_lag_data_iteration = 0;
        h_vertices_default_variable_edges_data_iteration = 0;
        h_vertices_default_variable_min_data_iteration = 0;
        h_vertices_default_variable_max_data_iteration = 0;
        h_vertices_default_variable_need_test_data_iteration = 0;
        h_vertices_default_variable_correct_data_iteration = 0;
        

	}
}


/*  Analytics Functions */

int reduce_vertice_default_id_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->id),  thrust::device_pointer_cast(d_vertices_default->id) + h_xmachine_memory_vertice_default_count);
}

int count_vertice_default_id_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_vertices_default->id),  thrust::device_pointer_cast(d_vertices_default->id) + h_xmachine_memory_vertice_default_count, count_value);
}
int min_vertice_default_id_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->id);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_vertice_default_id_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->id);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float reduce_vertice_default_value_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->value),  thrust::device_pointer_cast(d_vertices_default->value) + h_xmachine_memory_vertice_default_count);
}

float min_vertice_default_value_variable(){
    //min in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->value);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float max_vertice_default_value_variable(){
    //max in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->value);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float reduce_vertice_default_add_value_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->add_value),  thrust::device_pointer_cast(d_vertices_default->add_value) + h_xmachine_memory_vertice_default_count);
}

float min_vertice_default_add_value_variable(){
    //min in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->add_value);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float max_vertice_default_add_value_variable(){
    //max in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->add_value);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float reduce_vertice_default_previous_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->previous),  thrust::device_pointer_cast(d_vertices_default->previous) + h_xmachine_memory_vertice_default_count);
}

float min_vertice_default_previous_variable(){
    //min in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->previous);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float max_vertice_default_previous_variable(){
    //max in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->previous);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int reduce_vertice_default_max_lag_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->max_lag),  thrust::device_pointer_cast(d_vertices_default->max_lag) + h_xmachine_memory_vertice_default_count);
}

int count_vertice_default_max_lag_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_vertices_default->max_lag),  thrust::device_pointer_cast(d_vertices_default->max_lag) + h_xmachine_memory_vertice_default_count, count_value);
}
int min_vertice_default_max_lag_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->max_lag);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_vertice_default_max_lag_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->max_lag);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int reduce_vertice_default_current_lag_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->current_lag),  thrust::device_pointer_cast(d_vertices_default->current_lag) + h_xmachine_memory_vertice_default_count);
}

int count_vertice_default_current_lag_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_vertices_default->current_lag),  thrust::device_pointer_cast(d_vertices_default->current_lag) + h_xmachine_memory_vertice_default_count, count_value);
}
int min_vertice_default_current_lag_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->current_lag);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_vertice_default_current_lag_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->current_lag);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float reduce_vertice_default_min_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->min),  thrust::device_pointer_cast(d_vertices_default->min) + h_xmachine_memory_vertice_default_count);
}

float min_vertice_default_min_variable(){
    //min in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->min);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float max_vertice_default_min_variable(){
    //max in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->min);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float reduce_vertice_default_max_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->max),  thrust::device_pointer_cast(d_vertices_default->max) + h_xmachine_memory_vertice_default_count);
}

float min_vertice_default_max_variable(){
    //min in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->max);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
float max_vertice_default_max_variable(){
    //max in default stream
    thrust::device_ptr<float> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->max);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int reduce_vertice_default_need_test_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->need_test),  thrust::device_pointer_cast(d_vertices_default->need_test) + h_xmachine_memory_vertice_default_count);
}

int count_vertice_default_need_test_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_vertices_default->need_test),  thrust::device_pointer_cast(d_vertices_default->need_test) + h_xmachine_memory_vertice_default_count, count_value);
}
int min_vertice_default_need_test_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->need_test);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_vertice_default_need_test_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->need_test);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int reduce_vertice_default_correct_variable(){
    //reduce in default stream
    return thrust::reduce(thrust::device_pointer_cast(d_vertices_default->correct),  thrust::device_pointer_cast(d_vertices_default->correct) + h_xmachine_memory_vertice_default_count);
}

int count_vertice_default_correct_variable(int count_value){
    //count in default stream
    return (int)thrust::count(thrust::device_pointer_cast(d_vertices_default->correct),  thrust::device_pointer_cast(d_vertices_default->correct) + h_xmachine_memory_vertice_default_count, count_value);
}
int min_vertice_default_correct_variable(){
    //min in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->correct);
    size_t result_offset = thrust::min_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}
int max_vertice_default_correct_variable(){
    //max in default stream
    thrust::device_ptr<int> thrust_ptr = thrust::device_pointer_cast(d_vertices_default->correct);
    size_t result_offset = thrust::max_element(thrust_ptr, thrust_ptr + h_xmachine_memory_vertice_default_count) - thrust_ptr;
    return *(thrust_ptr + result_offset);
}



/* Agent functions */


	
/* Shared memory size calculator for agent function */
int vertice_send_message_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  
	return sm_size;
}

/** vertice_send_message
 * Agent function prototype for send_message function of vertice agent
 */
void vertice_send_message(hipStream_t &stream){

    int sm_size;
    int blockSize;
    int minGridSize;
    int gridSize;
    int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_vertice_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_vertice_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_vertice_list* vertices_default_temp = d_vertices;
	d_vertices = d_vertices_default;
	d_vertices_default = vertices_default_temp;
	//set working count to current state count
	h_xmachine_memory_vertice_count = h_xmachine_memory_vertice_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_vertice_count), &h_xmachine_memory_vertice_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_vertice_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_vertice_default_count), &h_xmachine_memory_vertice_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	//CONTINUOUS AGENT CHECK FUNCTION OUTPUT BUFFERS FOR OUT OF BOUNDS
	if (h_message_send_local_count + h_xmachine_memory_vertice_count > xmachine_message_send_local_MAX){
		printf("Error: Buffer size of send_local message will be exceeded in function send_message\n");
		exit(EXIT_FAILURE);
	}
	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_send_message, vertice_send_message_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = vertice_send_message_sm_size(blockSize);
	
	
	
	//SET THE OUTPUT MESSAGE TYPE FOR CONTINUOUS AGENTS
	//Set the message_type for non partitioned, spatially partitioned and On-Graph Partitioned message outputs
	h_message_send_local_output_type = optional_message;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_message_send_local_output_type), &h_message_send_local_output_type, sizeof(int)));
	//message is optional so reset the swap
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, reset_send_local_swaps, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	reset_send_local_swaps<<<gridSize, blockSize, 0, stream>>>(d_send_locals); 
	gpuErrchkLaunch();
	
	
	//MAIN XMACHINE FUNCTION CALL (send_message)
	//Reallocate   : false
	//Input        : 
	//Output       : send_local
	//Agent Output : 
	GPUFLAME_send_message<<<g, b, sm_size, stream>>>(d_vertices, d_send_locals, d_rand48);
	gpuErrchkLaunch();
	
	
	//CONTINUOUS AGENTS SCATTER NON PARTITIONED OPTIONAL OUTPUT MESSAGES
	//send_local Message Type Prefix Sum
	
	//swap output
	xmachine_message_send_local_list* d_send_locals_scanswap_temp = d_send_locals;
	d_send_locals = d_send_locals_swap;
	d_send_locals_swap = d_send_locals_scanswap_temp;
	
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_scan_storage_vertice, 
        temp_scan_storage_bytes_vertice, 
        d_send_locals_swap->_scan_input,
        d_send_locals_swap->_position,
        h_xmachine_memory_vertice_count, 
        stream
    );

	//Scatter
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, scatter_optional_send_local_messages, no_sm, state_list_size); 
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	scatter_optional_send_local_messages<<<gridSize, blockSize, 0, stream>>>(d_send_locals, d_send_locals_swap);
	gpuErrchkLaunch();
	
	//UPDATE MESSAGE COUNTS FOR CONTINUOUS AGENTS WITH NON PARTITIONED MESSAGE OUTPUT 
	gpuErrchk( hipMemcpy( &scan_last_sum, &d_send_locals_swap->_position[h_xmachine_memory_vertice_count-1], sizeof(int), hipMemcpyDeviceToHost));
	gpuErrchk( hipMemcpy( &scan_last_included, &d_send_locals_swap->_scan_input[h_xmachine_memory_vertice_count-1], sizeof(int), hipMemcpyDeviceToHost));
	//If last item in prefix sum was 1 then increase its index to get the count
	if (scan_last_included == 1){
		h_message_send_local_count += scan_last_sum+1;
	}else{
		h_message_send_local_count += scan_last_sum;
	}
    //Copy count to device
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_message_send_local_count), &h_message_send_local_count, sizeof(int)));	
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_vertice_default_count+h_xmachine_memory_vertice_count > xmachine_memory_vertice_MAX){
		printf("Error: Buffer size of send_message agents in state default will be exceeded moving working agents to next state in function send_message\n");
      exit(EXIT_FAILURE);
      }
      
  //pointer swap the updated data
  vertices_default_temp = d_vertices;
  d_vertices = d_vertices_default;
  d_vertices_default = vertices_default_temp;
        
	//update new state agent size
	h_xmachine_memory_vertice_default_count += h_xmachine_memory_vertice_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_vertice_default_count), &h_xmachine_memory_vertice_default_count, sizeof(int)));	
	
	
}



	
/* Shared memory size calculator for agent function */
int vertice_read_message_sm_size(int blockSize){
	int sm_size;
	sm_size = SM_START;
  //Continuous agent and message input has no partitioning
	sm_size += (blockSize * sizeof(xmachine_message_send_local));
	
	//all continuous agent types require single 32bit word per thread offset (to avoid sm bank conflicts)
	sm_size += (blockSize * PADDING);
	
	return sm_size;
}

/** vertice_read_message
 * Agent function prototype for read_message function of vertice agent
 */
void vertice_read_message(hipStream_t &stream){

    int sm_size;
    int blockSize;
    int minGridSize;
    int gridSize;
    int state_list_size;
	dim3 g; //grid for agent func
	dim3 b; //block for agent func

	
	//CHECK THE CURRENT STATE LIST COUNT IS NOT EQUAL TO 0
	
	if (h_xmachine_memory_vertice_default_count == 0)
	{
		return;
	}
	
	
	//SET SM size to 0 and save state list size for occupancy calculations
	sm_size = SM_START;
	state_list_size = h_xmachine_memory_vertice_default_count;

	

	//******************************** AGENT FUNCTION CONDITION *********************
	//THERE IS NOT A FUNCTION CONDITION
	//currentState maps to working list
	xmachine_memory_vertice_list* vertices_default_temp = d_vertices;
	d_vertices = d_vertices_default;
	d_vertices_default = vertices_default_temp;
	//set working count to current state count
	h_xmachine_memory_vertice_count = h_xmachine_memory_vertice_default_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_vertice_count), &h_xmachine_memory_vertice_count, sizeof(int)));	
	//set current state count to 0
	h_xmachine_memory_vertice_default_count = 0;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_vertice_default_count), &h_xmachine_memory_vertice_default_count, sizeof(int)));	
	
 

	//******************************** AGENT FUNCTION *******************************

	
	
	//calculate the grid block size for main agent function
	hipOccupancyMaxPotentialBlockSizeVariableSMem( &minGridSize, &blockSize, GPUFLAME_read_message, vertice_read_message_sm_size, state_list_size);
	gridSize = (state_list_size + blockSize - 1) / blockSize;
	b.x = blockSize;
	g.x = gridSize;
	
	sm_size = vertice_read_message_sm_size(blockSize);
	
	
	
	//BIND APPROPRIATE MESSAGE INPUT VARIABLES TO TEXTURES (to make use of the texture cache)
	
	
	//MAIN XMACHINE FUNCTION CALL (read_message)
	//Reallocate   : false
	//Input        : send_local
	//Output       : 
	//Agent Output : 
	GPUFLAME_read_message<<<g, b, sm_size, stream>>>(d_vertices, d_send_locals, d_rand48);
	gpuErrchkLaunch();
	
	
	//UNBIND MESSAGE INPUT VARIABLE TEXTURES
	
	
	//************************ MOVE AGENTS TO NEXT STATE ****************************
    
	//check the working agents wont exceed the buffer size in the new state list
	if (h_xmachine_memory_vertice_default_count+h_xmachine_memory_vertice_count > xmachine_memory_vertice_MAX){
		printf("Error: Buffer size of read_message agents in state default will be exceeded moving working agents to next state in function read_message\n");
      exit(EXIT_FAILURE);
      }
      
  //pointer swap the updated data
  vertices_default_temp = d_vertices;
  d_vertices = d_vertices_default;
  d_vertices_default = vertices_default_temp;
        
	//update new state agent size
	h_xmachine_memory_vertice_default_count += h_xmachine_memory_vertice_count;
	gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL( d_xmachine_memory_vertice_default_count), &h_xmachine_memory_vertice_default_count, sizeof(int)));	
	
	
}


 
extern void reset_vertice_default_count()
{
    h_xmachine_memory_vertice_default_count = 0;
}
